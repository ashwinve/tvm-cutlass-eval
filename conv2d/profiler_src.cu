
#include <iostream>
#include <chrono>
#include "cutlass/cutlass.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"

#define CUTLASS_CHECK(status)                                                                      {                                                                                                  cutlass::Status error = status;                                                                  if (error != cutlass::Status::kSuccess) {                                                          std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) << " at: " << __LINE__                 << std::endl;                                                                          throw std::runtime_error("cutlass error");                                                     }                                                                                              }


  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm0 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm1 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm2 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm3 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_256x128_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm4 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm5 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm6 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm7 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x256_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm8 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm9 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm10 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm11 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x128_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm12 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm13 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm14 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm15 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm16 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm17 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm18 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm19 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_128x64_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm20 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm21 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm22 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm23 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x64_32x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align8"
  using cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm24 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align4"
  using cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm25 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align2"
  using cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm26 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align1"
  using cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align1 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm75,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 8>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      1,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    2,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
  >::Kernel;

using ImplicitGemm27 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h1688fprop_optimized_64x128_64x2_nhwc_align1>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm28 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm29 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm30 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_32x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm31 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm32 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm33 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_32x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm34 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm35 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm36 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_32x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm37 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm38 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm39 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_32x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm40 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm41 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm42 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm43 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm44 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm45 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm46 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm47 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm48 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_32x5_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm49 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm50 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm51 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_32x6_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm52 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm53 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    6,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm54 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_32x6_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    10,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm55 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    10,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm56 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    10,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm57 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_32x10_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm58 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm59 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm60 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x128_64x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm61 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm62 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm63 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x256_64x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm64 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm65 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm66 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_256x64_64x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm67 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm68 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm69 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x256_64x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm70 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm71 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    4,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm72 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x128_64x4_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<64, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm73 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<64, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm74 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<64, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm75 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_128x64_64x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm76 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm77 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 64, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm78 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x128_64x3_nhwc_align2>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align8"
  using cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align8 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using ImplicitGemm79 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align8>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align4"
  using cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align4 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      4,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    4,
    4
  >::Kernel;

using ImplicitGemm80 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align4>;

  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align2"
  using cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align2 =
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::layout::TensorNHWC,
    cutlass::half_t,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64 >,
    cutlass::gemm::GemmShape<16, 8, 16>,

    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      2,
      cutlass::half_t,
      cutlass::half_t
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    2,
    2
  >::Kernel;

using ImplicitGemm81 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_h16816fprop_optimized_64x64_64x5_nhwc_align2>;

using ElementOutput = typename ImplicitGemm0::ElementC;
using ElementInputA = typename ImplicitGemm0::ElementA;
using ElementInputB = typename ImplicitGemm0::ElementB;
using ElementComputeEpilogue = typename ImplicitGemm0::ElementCompute;
using HostTensorA = cutlass::HostTensor<ElementInputA, typename ImplicitGemm0::LayoutA>;
using HostTensorB = cutlass::HostTensor<ElementInputB, typename ImplicitGemm0::LayoutB>;
using HostTensorC = cutlass::HostTensor<ElementOutput, typename ImplicitGemm0::LayoutC>;

struct Options {
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;

  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);
    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());
    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    int pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w;
    cmd.get_cmd_line_argument("pad_h", pad_h);
    cmd.get_cmd_line_argument("pad_w", pad_w);
    cmd.get_cmd_line_argument("stride_h", stride_h);
    cmd.get_cmd_line_argument("stride_w", stride_w);
    cmd.get_cmd_line_argument("dilation_h", dilation_h);
    cmd.get_cmd_line_argument("dilation_w", dilation_w);
    filter_size.c() = input_size.c();
    padding = {pad_h, pad_h, pad_w, pad_w};
    conv_stride = {stride_h, stride_w};
    dilation = {dilation_h, dilation_w};
  }

  cutlass::Tensor4DCoord output_size() const {
    auto dilated_h = (filter_size.h() - 1) * dilation.row() + 1;
    auto dilated_w = (filter_size.w() - 1) * dilation.column() + 1;
    auto h = (input_size.h() + padding.n() + padding.h() - dilated_h) / conv_stride.row() + 1;
    auto w = (input_size.w() + padding.w() + padding.c() - dilated_w) / conv_stride.column() + 1;
    return cutlass::Tensor4DCoord(input_size.n(), h, w, filter_size.n());
  }
};

template <typename ImplicitGemm>
double profile(ImplicitGemm implicit_gemm_op,
	       const cutlass::conv::Conv2dProblemSize& problem_size,
	       HostTensorA tensor_a,
	       HostTensorB tensor_b,
	       HostTensorC tensor_c,
	       HostTensorC tensor_ref_c,
	       hipEvent_t events[2],
	       cutlass::device_memory::allocation<uint8_t> workspace) {
  typename ImplicitGemm::Arguments arguments{
    problem_size,
      tensor_a.device_ref(),
      tensor_b.device_ref(),
      tensor_c.device_ref(),
      tensor_c.device_ref(),
      {ElementComputeEpilogue(1), ElementComputeEpilogue(0)},
      };

  auto status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);

  status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);
  status = implicit_gemm_op();
  CUTLASS_CHECK(status);

  hipEventRecord(events[0]);

  for (int iteration = 0; iteration < 100; ++iteration) {
    auto status = implicit_gemm_op();
    CUTLASS_CHECK(status);
  }

  hipEventRecord(events[1]);
  hipEventSynchronize(events[1]);
  float runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);

  return double(runtime_ms) / 100.0;
}

template <typename ImplicitGemm>
double try_profile(ImplicitGemm implicit_gemm_op,
		   const cutlass::conv::Conv2dProblemSize& problem_size,
		   HostTensorA tensor_a,
		   HostTensorB tensor_b,
		   HostTensorC tensor_c,
		   HostTensorC tensor_ref_c,
		   hipEvent_t events[2],
		   cutlass::device_memory::allocation<uint8_t> workspace) {
  try {
    using namespace std::chrono;
    //high_resolution_clock::time_point t1 = high_resolution_clock::now();
    auto t1 = system_clock::now();
    auto ret = profile(implicit_gemm_op, problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace);
    //high_resolution_clock::time_point t2 = high_resolution_clock::now();
    auto t2 = system_clock::now();
    duration<double> time_span = duration_cast<milliseconds>(t2 - t1);
    std::cout << ret << ", " << time_span.count() << std::endl;
    return ret;
  } catch(std::exception&) {
    return 999999;
  }
}

std::vector<double> profile_convolution(Options const &options) {
  auto oshape = options.output_size();
  HostTensorA tensor_a(options.input_size);
  HostTensorB tensor_b(options.filter_size);
  HostTensorC tensor_c(oshape);
  HostTensorC tensor_ref_c(oshape);

  cutlass::conv::Conv2dProblemSize problem_size(
						options.input_size,
						options.filter_size,
						options.padding,
						options.conv_stride,
						options.dilation,
						options.output_size(),
						cutlass::conv::Mode::kCrossCorrelation,
						1
						);

  hipEvent_t events[2];
  for (auto & event : events) {
    hipEventCreate(&event);
  }

  ImplicitGemm0 conv2d_op;
  typename ImplicitGemm0::Arguments arguments{
    problem_size,
      tensor_a.device_ref(),
      tensor_b.device_ref(),
      tensor_c.device_ref(),
      tensor_c.device_ref(),
      {ElementComputeEpilogue(1), ElementComputeEpilogue(0)},
      };

  size_t workspace_size = conv2d_op.get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  std::vector<double> elapsed;
  elapsed.push_back(try_profile(ImplicitGemm0(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm1(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm2(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm3(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm4(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm5(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm6(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm7(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm8(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm9(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm10(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm11(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm12(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm13(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm14(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm15(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm16(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm17(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm18(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm19(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm20(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm21(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm22(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm23(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm24(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm25(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm26(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm27(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm28(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm29(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm30(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm31(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm32(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm33(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm34(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm35(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm36(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm37(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm38(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm39(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm40(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm41(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm42(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm43(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm44(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm45(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm46(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm47(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm48(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm49(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm50(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm51(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm52(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm53(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm54(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm55(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm56(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm57(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm58(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm59(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm60(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm61(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm62(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm63(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm64(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm65(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm66(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm67(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm68(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm69(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm70(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm71(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm72(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm73(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm74(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm75(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm76(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm77(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm78(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm79(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm80(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));
  elapsed.push_back(try_profile(ImplicitGemm81(), problem_size, tensor_a, tensor_b, tensor_c, tensor_ref_c, events, workspace));

  for (auto event : events) {
    (void)hipEventDestroy(event);
  }

  return elapsed;
}

int main(int argc, char const **args) {
  Options options;
  options.parse(argc, args);
  auto elapsed = profile_convolution(options);

  double total = 0;
  for (auto t: elapsed) {
    //    std::cout << t << std::endl;
    if (t < 100) {
    total += t * 100.0;
    }
  }
  std::cout << "total: " << total << std::endl;
  return 0;
}